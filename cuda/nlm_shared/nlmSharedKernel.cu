
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

// Array access macros
#define INPUT(i,j) input_grid[(j) + (i)*(N)]
#define TEMP(i,j) temp_grid[(j) + (i)*(N)]

#define WINDOW_SIZE (7)
#define NEIGHBOR_SIZE (3)
#define BLOCK_SIZE (512)

#define FILTER_SIZE ((WINDOW_SIZE) + (NEIGHBOR_SIZE) - 1)
#define FILTER_RADIUS (((FILTER_SIZE) - 1) / 2)

__global__ void nlmSimple(int N, double const *input_grid, double *output_grid, float filtSigma)
{
  __shared__ double temp_grid[BLOCK_SIZE * FILTER_SIZE];

  // Define global and local indices of current pixel
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x + FILTER_RADIUS * blockDim.x;

  int pix_ix, pix_iy, pix_jx, pix_jy;

  double neighbor_j,
         neighbor_i,
         output = 0,
         sum_weights = 0;

  // Read input elements into shared memory
  for (int i = -FILTER_RADIUS; i <= FILTER_RADIUS; i++)
  {
    if ((int)blockIdx.x + i >= 0 && (int)blockIdx.x + i < N)
    {
      temp_grid[lindex + i * (int)blockDim.x] = input_grid[gindex + i * (int)blockDim.x];
    }
  }

  // Synchronize (ensure all the data is available)
  __syncthreads();

  pix_iy = lindex % N;
  pix_ix = (lindex - pix_iy) / N;

  if (pix_ix < FILTER_SIZE && pix_iy < N)
  { 
    int window_radius = (WINDOW_SIZE - 1) / 2;
    int neighbor_radius = (NEIGHBOR_SIZE - 1) / 2; 

    // Iterate through window
    for (int k = -window_radius; k <= window_radius; k++)
      for (int l = -window_radius; l <= window_radius; l++)
      {
        double weight = 0;
        double distance = 0;

        pix_jx = pix_ix + k; 
        pix_jy = pix_iy + l;

        if (pix_jx < 0 || pix_jx >= FILTER_SIZE ||
            pix_jy < 0 || pix_jy >= N)
          continue;

        // Iterate through every pix_j neighbors
        for (int p = -neighbor_radius; p <= neighbor_radius; p++)
          for (int q = -neighbor_radius; q <= neighbor_radius; q++)
          {
            if (pix_jx + p < 0 || pix_jx + p >= FILTER_SIZE ||
                pix_jy + q < 0 || pix_jy + q >= N ||
                pix_ix + p < 0 || pix_ix + p >= FILTER_SIZE ||
                pix_iy + q < 0 || pix_iy + q >= N)
              continue;

            neighbor_j = TEMP(pix_jx + p, pix_jy + q);
            neighbor_i = TEMP(pix_ix + p, pix_iy + q);
            distance += (neighbor_i - neighbor_j) * (neighbor_i - neighbor_j);
          }

        // Derive weight for pixels i and j
        weight = __expf(-(distance / filtSigma + 
              (k*k + l*l) * (1.0f)/(float)(WINDOW_SIZE* WINDOW_SIZE)));

        sum_weights += weight;

        // Sum for every pixel in the window
        output += TEMP(pix_jx, pix_jy) * weight;			
      }

    // Normalize
    sum_weights = (double)(1 / sum_weights);
    output *= sum_weights;

    // Write output to global memory
    output_grid[gindex] = output;
  }
}
