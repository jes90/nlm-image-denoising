
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

// Array access macros
#define INPUT(i,j) input_grid[(j) + (i)*(N)]

#define WINDOW_SIZE (7)
#define NEIGHBOR_SIZE (3)

__global__ void nlmSimple(int N, double const *input_grid, double *output_grid, float filtSigma)
{
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;

	int pix_ix, 
		pix_iy, 
		pix_jx,
		pix_jy;

	double neighbor_j,
			neighbor_i,
			output = 0,
			sum_weights = 0;
	
	pix_iy = gindex % N;
	pix_ix = (gindex - pix_iy) / N;

	if (pix_ix < N && pix_iy < N)
	{ 
		int window_radius = (WINDOW_SIZE - 1) / 2;
		int neighbor_radius = (NEIGHBOR_SIZE - 1) / 2; 
			
		// Iterate through window
		for (int k = -window_radius; k <= window_radius; k++)
			for (int l = -window_radius; l <= window_radius; l++)
			{
				double weight = 0;
				double distance = 0;

				pix_jx = pix_ix + k; 
				pix_jy = pix_iy + l;

				if (pix_jx < 0 || pix_jx >= N ||
					pix_jy < 0 || pix_jy >= N)
					continue;

				// Iterate through every pix_j neighbors
				for (int p = -neighbor_radius; p <= neighbor_radius; p++)
					for (int q = -neighbor_radius; q <= neighbor_radius; q++)
					{
						if (pix_jx + p < 0 || pix_jx + p >= N ||
							pix_jy + q < 0 || pix_jy + q >= N ||
							pix_ix + p < 0 || pix_ix + p >= N ||
							pix_iy + q < 0 || pix_iy + q >= N)
							continue;
						
						neighbor_j = INPUT(pix_jx + p, pix_jy + q);
						neighbor_i = INPUT(pix_ix + p, pix_iy + q);
						distance += (neighbor_i - neighbor_j) * (neighbor_i - neighbor_j);
					}

				// Derive weight for pixels i and j
				weight = __expf(-(distance / filtSigma + 
								(k*k + l*l) * (1.0f)/(float)(WINDOW_SIZE* WINDOW_SIZE)));

				sum_weights += weight;
				
				// Sum for every pixel in the window
				output += INPUT(pix_jx, pix_jy) * weight;				
			}

		// Normalize
		sum_weights = (double)(1 / sum_weights);
		output *= sum_weights;

		// Write output to global memory
		output_grid[gindex] = output;
	}
}
